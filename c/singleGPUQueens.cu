#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <sys/time.h>
#define _QUEENS_BLOCK_SIZE_ 	128
#define _EMPTY_      -1



double rtclock()
{
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday (&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d",stat);
    return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}



typedef struct queen_root{
    unsigned int control;
    int8_t board[12]; //maximum depth of the solution space.
} QueenRoot;


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


inline void prefixesHandleSol(QueenRoot *root_prefixes,unsigned int flag,char *board,int initialDepth,int num_sol){

    root_prefixes[num_sol].control = flag;

    for(int i = 0; i<initialDepth;++i)
      root_prefixes[num_sol].board[i] = (char)board[i];
}

inline bool MCstillLegal(const char *board, const int r)
{

    int i;
    int ld;
    int rd;
  // Check vertical
  for ( i = 0; i < r; ++i)
    if (board[i] == board[r]) return false;
    // Check diagonals
    ld = board[r];  //left diagonal columns
    rd = board[r];  // right diagonal columns
    for ( i = r-1; i >= 0; --i) {
      --ld; ++rd;
      if (board[i] == ld || board[i] == rd) return false;
    }

    return true;
}

__device__  bool GPU_queens_stillLegal(const char *board, const int r){

  bool safe = true;
  int i;
  register int ld;
  register int rd;
  // Check vertical
  for ( i = 0; i < r; ++i)
    if (board[i] == board[r]) safe = false;
    // Check diagonals
    ld = board[r];  //left diagonal columns
    rd = board[r];  // right diagonal columns
    for ( i = r-1; i >= 0; --i) {
      --ld; ++rd;
      if (board[i] == ld || board[i] == rd) safe = false;
    }

    return safe;
}


__global__ void BP_queens_root_dfs(int N, unsigned int nPrefixes, int initial_depth,
    QueenRoot *root_prefixes,unsigned long long int *vector_of_tree_size, unsigned long long int *sols){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < nPrefixes) {
        unsigned int flag = 0;
        unsigned int bit_test = 0;
        char board[32]; //representa o ciclo
        int N_l = N;
        int i, depth;
        unsigned long long  qtd_sols_thread = 0ULL;
        int depthGlobal = initial_depth;
        unsigned long long int tree_size = 0ULL;

        for (i = 0; i < N_l; ++i) {
            board[i] = _EMPTY_;
        }

        flag = root_prefixes[idx].control;


        for (i = 0; i < depthGlobal; ++i)
            board[i] = root_prefixes[idx].board[i];

        depth=depthGlobal;

        do{

            board[depth]++;
            bit_test = 0;
            bit_test |= (1<<board[depth]);

            if(board[depth] == N_l){
                board[depth] = _EMPTY_;
                //if(block_ub > upper)   block_ub = upper;
            }else if (!(flag &  bit_test ) && GPU_queens_stillLegal(board, depth)){

                    ++tree_size;
                    flag |= (1ULL<<board[depth]);

                    depth++;

                    if (depth == N_l) { //sol
                        ++qtd_sols_thread ;
                    }else continue;
                }else continue;

            depth--;
            flag &= ~(1ULL<<board[depth]);

            }while(depth >= depthGlobal); //FIM DO DFS_BNB

        sols[idx] = qtd_sols_thread ;
        vector_of_tree_size[idx] = tree_size;
    }//if
}//kernel
////////



unsigned long long int BP_queens_prefixes(int size, int initialDepth ,unsigned long long *tree_size, QueenRoot *root_prefixes){

    unsigned int flag = 0;
    int bit_test = 0;
    char board[32]; //representa o ciclo
    int i, depth; //para dizer que 0-1 ja foi visitado e a busca comeca de 1, bote 2
    unsigned long long int local_tree = 0ULL;
    unsigned long long int num_sol = 0;
   //register int custo = 0;

    /*initialization*/
    for (i = 0; i < size; ++i) { //
        board[i] = -1;
    }

    depth = 0;

    do{

        board[depth]++;
        bit_test = 0;
        bit_test |= (1<<board[depth]);


        if(board[depth] == size){
            board[depth] = _EMPTY_;
                //if(block_ub > upper)   block_ub = upper;
        }else if ( MCstillLegal(board, depth) && !(flag &  bit_test ) ){ //is legal

                flag |= (1ULL<<board[depth]);
                depth++;
                ++local_tree;
                if (depth == initialDepth){ //handle solution
                   prefixesHandleSol(root_prefixes,flag,board,initialDepth,num_sol);
                   num_sol++;
            }else continue;
        }else continue;

        depth--;
        flag &= ~(1ULL<<board[depth]);

    }while(depth >= 0);

    *tree_size = local_tree;

    return num_sol;
}

void GPU_call_cuda_queens(int size, int initial_depth, int block_size,unsigned int n_explorers, QueenRoot *root_prefixes_h ,
	unsigned long long int *vector_of_tree_size_h, unsigned long long int *sols_h){



    unsigned long long int *vector_of_tree_size_d;
    unsigned long long int *sols_d;
    QueenRoot *root_prefixes_d;

    int num_blocks = ceil((double)n_explorers/block_size);


    hipMalloc((void**) &vector_of_tree_size_d,n_explorers*sizeof(unsigned long long int));
    hipMalloc((void**) &sols_d,n_explorers*sizeof(unsigned long long int));
    hipMalloc((void**) &root_prefixes_d,n_explorers*sizeof(QueenRoot));

    //I Think this is not possible in Chapel. It must be internal
    hipMemcpy(root_prefixes_d, root_prefixes_h, n_explorers * sizeof(QueenRoot), hipMemcpyHostToDevice);

    printf("\n### Regular BP-DFS search. ###\n");

    //kernel_start =  rtclock();


    BP_queens_root_dfs<<< num_blocks,block_size>>> (size,n_explorers,initial_depth,root_prefixes_d, vector_of_tree_size_d,sols_d);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    //kernel_stop = rtclock();

    hipMemcpy(vector_of_tree_size_h,vector_of_tree_size_d,n_explorers*sizeof(unsigned long long int),hipMemcpyDeviceToHost);
    hipMemcpy(sols_h,sols_d,n_explorers*sizeof(unsigned long long int),hipMemcpyDeviceToHost);

    hipFree(vector_of_tree_size_d);
    hipFree(sols_d);
    hipFree(root_prefixes_d);



    //After that, Chapel reduces the values
}


double call_queens(int size, int initialDepth, int block_size){


    unsigned long long initial_tree_size = 0ULL;
    unsigned long long qtd_sols_global = 0ULL;
    unsigned long long gpu_tree_size = 0ULL;

    unsigned int nMaxPrefixes = 75580635;



    printf("\n### Queens size: %d, Initial depth: %d, Block size: %d", initialDepth, size, block_size);

    double initial_time = rtclock();

    QueenRoot* root_prefixes_h = (QueenRoot*)malloc(sizeof(QueenRoot)*nMaxPrefixes);
    unsigned long long int *vector_of_tree_size_h = (unsigned long long int*)malloc(sizeof(unsigned long long int)*nMaxPrefixes);
    unsigned long long int *solutions_h = (unsigned long long int*)malloc(sizeof(unsigned long long int)*nMaxPrefixes);

    //initial search, getting the tree root nodes for the gpu;
    unsigned long long n_explorers = BP_queens_prefixes((short)size, initialDepth ,&initial_tree_size, root_prefixes_h);

    //calling the gpu-based search
    GPU_call_cuda_queens(size, initialDepth, block_size, n_explorers, root_prefixes_h ,vector_of_tree_size_h, solutions_h);

    printf("\nInitial tree size: %llu", initial_tree_size );

    double final_time = rtclock();

    for(int i = 0; i<n_explorers;++i){
        if(solutions_h[i]>0)
            qtd_sols_global += solutions_h[i];
        if(vector_of_tree_size_h[i]>0)
            gpu_tree_size +=vector_of_tree_size_h[i];

    }

    printf("\nGPU Tree size: %llu\nTotal tree size: %llu\nNumber of solutions found: %llu.\n", gpu_tree_size,(initial_tree_size+gpu_tree_size),qtd_sols_global );
    printf("\nElapsed total: %.3f\n", (final_time-initial_time));

    return (final_time-initial_time);
}


int main(int argc, char *argv[]){

    int block_size;
    int initialDepth;
    int size;

  block_size =   atoi(argv[3]);
    initialDepth = atoi(argv[2]);
    size = atoi(argv[1]);

    call_queens(size, initialDepth, block_size);



    return 0;
}
